
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void add_vector(int *a, int *b, int *c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;    
    c[i] = a[i] + b[i];
}

int main(void)
{
    int *a, *b, *c;

    // Allocate unified memory
    hipMallocManaged(&a, N * sizeof(int));
    hipMallocManaged(&b, N * sizeof(int));
    hipMallocManaged(&c, N * sizeof(int));

    // Initialize the input arrays
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = N - 1 - i;
    }

    // Launch the kernel
    add_vector<<<2, 5>>>(a, b, c);

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    // Print the result
    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}
