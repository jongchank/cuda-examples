
#include <hip/hip_runtime.h>
#include <stdio.h>

#define X 10
#define Y 10

__global__ void add_matrix(int *a, int *b, int *c)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    c[x * Y + y] = a[x * Y + y] + b[x * Y + y];
}

int main(void)
{
    int a[X][Y] = {
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9}
    };
    int b[X][Y] = {
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9}
    };
    int c[X][Y];
    int *d_a, *d_b, *d_c;
    int i;

    hipMalloc((void **)&d_a, X * Y * sizeof(int));
    hipMalloc((void **)&d_b, X * Y * sizeof(int));
    hipMalloc((void **)&d_c, X * Y * sizeof(int));

    hipMemcpy(d_a, a, X * Y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, X * Y * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(5, 5);
    dim3 dimGrid(2, 2);
    add_matrix<<<dimBlock, dimGrid>>>(d_a, d_b, d_c);

    hipDeviceSynchronize();

    hipMemcpy(c, d_c, X * Y * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for (i = 0; i < X; i++) {
        for (int j = 0; j < Y; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    return 0;
}
