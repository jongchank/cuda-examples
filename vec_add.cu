
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void add_vector(int *a, int *b, int *c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;    
    c[i] = a[i] + b[i];
}

int main(void)
{
    int a[N] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    int b[N] = {9, 8, 7, 6, 5, 4, 3, 2, 1, 0};
    int c[N];
    int *d_a, *d_b, *d_c;
    int i;

    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add_vector<<<2, 5>>>(d_a, d_b, d_c);

    hipDeviceSynchronize();

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for (i = 0; i < N; i++) {
        printf("%d\n", c[i]);
    }

    return 0;
}
