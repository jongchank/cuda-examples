
#include <hip/hip_runtime.h>
#include <stdio.h>

#define X 10
#define Y 10

__global__ void add_matrix(int *a, int *b, int *c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;    
    c[i] = a[i] + b[i];
}

int main(void)
{
    int a[X][Y] = {
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9}
    };
    int b[X][Y] = {
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
        {0, 1, 2, 3, 4, 5, 6, 7, 8, 9}
    };
    int c[X][Y];
    int *d_a, *d_b, *d_c;
    int i;

    hipMalloc((void **)&d_a, X * Y * sizeof(int));
    hipMalloc((void **)&d_b, X * Y * sizeof(int));
    hipMalloc((void **)&d_c, X * Y * sizeof(int));

    hipMemcpy(d_a, a, X * Y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, X * Y * sizeof(int), hipMemcpyHostToDevice);

    add_matrix<<<10, 10>>>(d_a, d_b, d_c);

    hipDeviceSynchronize();

    hipMemcpy(c, d_c, X * Y * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for (i = 0; i < X; i++) {
        for (int j = 0; j < Y; j++) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    return 0;
}
